
#include "hip/hip_runtime.h"
#include "../cuda_struct.h"
#include ""
#include <stdio.h>
#include "../FFxtubes.h"


Systdata Syst1a,Syst2a;


int main()
{
	// Idea will be:

	// . Load systdata

	Syst1a.InvokeHost(NUMBER_OF_VERTICES_AIMED);
	Syst2a.InvokeHost(NUMBER_OF_VERTICES_AIMED);
	
	Syst1a.LoadHost("testsyst.sdt");	
	
	// . Call PerformAdvance:
	// This expects systdata populated on host memory.
	
	PerformCUDA_Advance_2 (
		&Syst1a, 
		Syst1a.Nverts,
		1e-13, 
		10,
		&Syst2a);
	
	Syst2a.AsciiOutput("output.txt");
	
	printf("done.");
	
	Syst1a.RevokeHost();
	Syst2a.RevokeHost();

	getch();

	return 0;
}
