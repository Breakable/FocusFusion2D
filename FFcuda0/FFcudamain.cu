
#include "hip/hip_runtime.h"
#include "../cuda_struct.h"
#include ""
#include <stdio.h>


Systdata Syst1,Syst2;


int main()
{
	// Idea will be:

	// . Load systdata

	Syst1.InvokeHost(NUMBER_OF_VERTICES);
	Syst2.InvokeHost(NUMBER_OF_VERTICES);
	
	Syst1.LoadHost("testsyst.sdt");	
	
	// . Call PerformAdvance:
	// This expects systdata populated on host memory.
	
	PerformCUDA_Advance_2 (
		&Syst1, 
		Syst1.Nverts,
		1e-13, 
		10,
		&Syst2,
		f64 t // time of first timeslice
		);
	
	Syst2.AsciiOutput("output.txt");
	
	printf("done.");
	
	Syst1.RevokeHost();
	Syst2.RevokeHost();

	getch();

	return 0;
}
